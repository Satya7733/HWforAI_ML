#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <chrono>
#include <fstream>

// Network & benchmark params
constexpr int BATCH = 128;
constexpr int IN    = 4;
constexpr int H     = 5;
constexpr int OUT   = 1;
constexpr int STEPS = 1000;

// CUDA error checking
#define CUDA_CHECK(call)                                                     \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n";         \
      std::exit(1);                                                           \
    }                                                                         \
  } while (0)

// cuBLAS error checking
#define CUBLAS_CHECK(call)                                                   \
  do {                                                                        \
    hipblasStatus_t st = call;                                                \
    if (st != HIPBLAS_STATUS_SUCCESS) {                                        \
      std::cerr << "cuBLAS error\n";                                          \
      std::exit(1);                                                           \
    }                                                                         \
  } while (0)

// Adds bias + ReLU in-place
__global__ void add_bias_relu(float* x, const float* bias, int rows, int cols) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < rows*cols) {
    int j = idx % cols;
    float v = x[idx] + bias[j];
    x[idx] = v > 0 ? v : 0;
  }
}

int main(){
  // 1) Allocate & init host data
  std::vector<float> h_x(BATCH*IN), h_W1(IN*H), h_b1(H),
                     h_W2(H*OUT), h_b2(OUT);
  srand(0);
  auto rnd = [](float& v){ v = rand()/float(RAND_MAX)-0.5f; };
  for (auto& v : h_x)  rnd(v);
  for (auto& v : h_W1) rnd(v);
  for (auto& v : h_b1) rnd(v);
  for (auto& v : h_W2) rnd(v);
  for (auto& v : h_b2) rnd(v);

  // 2) Allocate device buffers
  float *d_x, *d_y1, *d_y2, *d_W1, *d_b1, *d_W2, *d_b2;
  CUDA_CHECK(hipMalloc(&d_x,  BATCH*IN*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_y1, BATCH*H*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_y2, BATCH*OUT*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_W1, IN*H*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_b1, H*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_W2, H*OUT*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_b2, OUT*sizeof(float)));

  // 3) Copy params Host→Device
  CUDA_CHECK(hipMemcpy(d_x,  h_x.data(),  BATCH*IN*sizeof(float),  hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W1, h_W1.data(), IN*H*sizeof(float),    hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b1, h_b1.data(), H*sizeof(float),      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_W2, h_W2.data(), H*OUT*sizeof(float),   hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b2, h_b2.data(), OUT*sizeof(float),     hipMemcpyHostToDevice));

  // 4) cuBLAS handle
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));

  // Warm-up
  const float α = 1.0f, β = 0.0f;
  CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           H, BATCH, IN, &α,
                           d_W1, H, d_x, IN, &β,
                           d_y1, H));
  add_bias_relu<<<(BATCH*H+255)/256,256>>>(d_y1, d_b1, BATCH, H);
  CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           OUT, BATCH, H, &α,
                           d_W2, OUT, d_y1, H, &β,
                           d_y2, OUT));
  hipDeviceSynchronize();

  // Timed loop
  hipEvent_t t0, t1;
  CUDA_CHECK(hipEventCreate(&t0));
  CUDA_CHECK(hipEventCreate(&t1));
  CUDA_CHECK(hipEventRecord(t0));
  for(int i=0; i<STEPS; ++i){
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             H, BATCH, IN, &α,
                             d_W1, H, d_x, IN, &β,
                             d_y1, H));
    add_bias_relu<<<(BATCH*H+255)/256,256>>>(d_y1, d_b1, BATCH, H);
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             OUT, BATCH, H, &α,
                             d_W2, OUT, d_y1, H, &β,
                             d_y2, OUT));
  }
  CUDA_CHECK(hipEventRecord(t1));
  CUDA_CHECK(hipEventSynchronize(t1));

  float ms = 0.f;
  CUDA_CHECK(hipEventElapsedTime(&ms, t0, t1));

  // Write timing to CSV
  std::ofstream fout("cuda_times.csv", std::ios::app);
  fout << "IN4_H5_OUT1," << ms << "\n";
  fout.close();
  std::cout << "Custom CUDA forward time (ms): " << ms << "\n";

  // Cleanup
  hipblasDestroy(handle);
  hipFree(d_x);  hipFree(d_y1); hipFree(d_y2);
  hipFree(d_W1); hipFree(d_b1); hipFree(d_W2); hipFree(d_b2);
  return 0;
}
